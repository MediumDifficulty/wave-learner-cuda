#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "trainer.h"

// TODO: Research if numel is nescesarry

__device__ float frac(float x) {
    return x - floorf(x);
}

__device__ void fcarray_remove(FunctionCoefficients functions[], int index, unsigned int* length) {
    for (int i = index + 1; i < *length; i++) {
        functions[i - 1] = functions[i];
    }
    
    *length -= 1;
}

__device__ float array_to_local(float x, int goal_res) {
    return x / static_cast<float>(goal_res) * 2.f - 1.f;
}

__device__ float function_coefficients_eval(FunctionCoefficients* fc, float x) {
    switch (fc->function_type) {
        case Sine:
            return sin(x + fc->x_translation) * fc->scale;
        case SawTooth:
            return frac(x + fc->x_translation) * fc->scale;
    }
}

__device__ FunctionCoefficients function_coefficients_rand(hiprandState *hiprandState) {
    FunctionCoefficients fc;

    fc.function_type = static_cast<WaveFunction>(hiprand(hiprandState) % WaveFunction::Count);
    fc.scale = hiprand_uniform(hiprandState) * 2.f - 1.f;
    fc.x_translation = hiprand_uniform(hiprandState) * 2.f - 1.f;

    return fc;
}

__device__ Agent agent_rand(HyperParameters* params, hiprandState *hiprandState) {
    Agent agent;
    agent.fitness = 0.f;
    agent.functions_len = params->starting_functions;

    for (int i = 0; i < params->starting_functions; i++) {
        agent.functions[i] = function_coefficients_rand(hiprandState);
    }

    return agent;
};

__device__ float agent_evaluate(Agent* agent, float x) {
    float evaluation = 0.f;
    for (int i = 0; i < agent->functions_len; i++) {
        evaluation += function_coefficients_eval(&agent->functions[i], x);
    }

    return evaluation;
}

__device__ void agent_compute_fitness(Agent* agent, float *goal, int goal_res) {
    agent->fitness = 0.f;
    for (int i = 0; i < goal_res; i++) {
        float x = array_to_local(static_cast<float>(i), goal_res);
        agent->fitness -= abs(agent_evaluate(agent, x) - goal[i]);
    }
}

__device__ Agent agent_crossover(Agent* parent_a, Agent* parent_b, hiprandState *hiprandState) {
    int minimum = min(parent_a->functions_len, parent_b->functions_len);
    Agent agent;
    agent.fitness = 0.f;
    agent.functions_len = minimum;

    for (int i = 0; i < minimum; i++) {
        FunctionCoefficients* f1 = &parent_a->functions[i];
        FunctionCoefficients* f2 = &parent_b->functions[i];

        if (hiprand_uniform(hiprandState) > 0.5) {
            agent.functions[i] = *f1;
        } else {
            agent.functions[i] = *f2;
        }
    }

    return agent;
}

__device__ void agent_mutate(Agent* agent, HyperParameters* params, hiprandState *hiprandState) {
    if (agent->functions_len < MAX_FUNCTIONS && hiprand_uniform(hiprandState) < params->function_addition_probability) {
        agent->functions[agent->functions_len] = function_coefficients_rand(hiprandState);
        agent->functions_len++;
    }

    if (agent->functions_len > 1 && hiprand_uniform(hiprandState) < params->function_subtraction_probability) {
        fcarray_remove(agent->functions, hiprand(hiprandState) % agent->functions_len, &agent->functions_len);
    }

    for (int i = 0; i < agent->functions_len; i++) {
        if (hiprand_uniform(hiprandState) < params->mutation_probability) {
            agent->functions[i].scale += hiprand_normal(hiprandState) * params->mutation_strength;
        }

        if (hiprand_uniform(hiprandState) < params->mutation_probability) {
            agent->functions[i].x_translation += hiprand_normal(hiprandState) * params->mutation_strength;
        }
    }

    // TODO: Maybe mutate function type?
}

extern "C" __global__ void init_kernel(hiprandState *state, Agent *agents, HyperParameters params, float* goal, int goal_res, unsigned long long seed, int numel) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < numel) {    
        hiprand_init(seed, id, 0, &state[id]);

        agents[id] = agent_rand(&params, &state[id]);
        agent_compute_fitness(&agents[id], goal, goal_res);

        // agents[id].fitness = 5.f;
    }
}

extern "C" __global__ void step_kernel(hiprandState *state, Agent *agents, HyperParameters params, float* goal, int goal_res, int agents_len, int numel) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < numel) {
        int top = agents_len - numel;

        // TODO: `top +` is for debugging purposes only
        Agent* parent_a = &agents[hiprand(&state[id]) % top];
        Agent* parent_b = &agents[hiprand(&state[id]) % top];

        // Crossover
        Agent child = agent_crossover(parent_a, parent_b, &state[id]);

        // Mutation
        agent_mutate(&child, &params, &state[id]);
        // Evaluate
        agent_compute_fitness(&child, goal, goal_res);

        agents[id + top] = child;
        // agents[id] = child;
    }
}

// https://gist.github.com/mre/1392067
extern "C" __global__ void step_sort_kernel(Agent *agents, int j, int k, int numel) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < numel) {
        unsigned int ixj = i^j;

        // TODO: This might sort the wrong way backwards
        if (ixj > i) {
            if ((i & k) == 0) {
                if (agents[i].fitness < agents[ixj].fitness) {
                    Agent temp = agents[i];
                    agents[i] = agents[ixj];
                    agents[ixj] = temp;
                }
            }
            if ((i & k) != 0) {
                if (agents[i].fitness > agents[ixj].fitness) {
                    Agent temp = agents[i];
                    agents[i] = agents[ixj];
                    agents[ixj] = temp;
                }
            }
        }
    }
}

extern "C" __global__ void output_kernel(Agent *agents, float *buff, int goal_res, int index, int numel) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < numel) {
        buff[id] = agent_evaluate(&agents[index], array_to_local(static_cast<float>(id), goal_res));
    }
}